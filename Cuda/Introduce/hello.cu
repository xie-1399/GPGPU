


#include <hip/hip_runtime.h>
#include <stdio.h>
/* write the simple kernel function to print at GPU */

__global__ void helloFromGPU(void){
    /* the common block and thread representation */
    const int bid = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;

    const int tid = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    printf("Hello World from GPU block %d and thread %d !\n",bid,tid);

}


int main(){
    printf("Hello World from CPU!\n");
    dim3 grid(2,2,3);
    dim3 block(2,2,3);
    helloFromGPU<<<grid,block>>>(); /* 10 threads are called to execute the func */
    hipDeviceReset(); /* clear all resources about the Device */
    return 0;
}

/*
    Hello World from CPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
 */