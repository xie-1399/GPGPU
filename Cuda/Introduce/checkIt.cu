
/* this show a more complex shape of the cuda thread */
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void)
{
    printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d) gridDim(%d,%d,%d)\n",threadIdx.x,threadIdx.y,threadIdx.z,
           blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,
           gridDim.x,gridDim.y,gridDim.z);
}

void complexShape(){
    int nElem = 48;
    dim3 block(3,2,2);
    dim3 grid(2,2);
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
    checkIndex<<<grid,block>>>();
    hipDeviceSynchronize();
}

int main(int argc,char **argv)
{
    /* define two blocks and each block has three threads */
    int nElem=6;
    dim3 block(3);
    dim3 grid((nElem+block.x-1)/block.x);
    printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
    checkIndex<<<grid,block>>>();
    hipDeviceReset();
    complexShape(); /* a more complex shape trans */
    return 0;
}