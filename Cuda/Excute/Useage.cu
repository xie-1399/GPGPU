
#include <hip/hip_runtime.h>
#include <stdio.h>

/* here should understand more details in c++ */

/* return the value to the add*/
__device__ double add1(const double x , const double y){
    return (x + y);
}

__global__ void vecaddUse1(double *x, double *y, double *z,const int N){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n < N) { z[n] = add1(x[n],y[n]);}
}

/* use the pointer in the device */
__device__ void add2(double x , double y , double *z){
    *z = x + y;
}

__global__ void vecaddUse2(double *x, double *y, double *z,const int N){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n < N) {
        add2(x[n],y[n],&z[n]);
    }
}

/* use the ref in the device */
__device__ void add3(double x , double y , double &z){
    z = x + y;
}

__global__ void vecaddUse3(double *x, double *y, double *z,const int N){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n < N) {
        add3(x[n],y[n],z[n]);
    }
}
