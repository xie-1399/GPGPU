#include "hip/hip_runtime.h"
/* the vec add with cuda version */
#include <stdio.h>
#include <math.h>
#include "excute.cuh"
#include <iostream>
using namespace std;
const float diff = 1.0e-15;
const float a_value = 1.03f;
const float b_value = 1.02f;
const float c_value = 2.05f;

/* the kernel function to cal the vec add value  */
__global__ void vec_add(float* a,float* b,float* c,const int N){
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n > N) return;  /* can use the return to avoid some logic errors */
    c[n] = a[n] + b[n];
}

void check(const float *z,const int N){
    bool has_error = false;
    for(int i = 0; i < N;i++){
        if(fabs(z[i] - c_value) > diff){
            cout << (z[i] - c_value) << endl;
            has_error = true;
        }
    }
    printf("%s\n",has_error ? "Has Error": "Right ");
}

int main(){

    const int N = 10000;
    const int size = sizeof (float) * N;
    float *x_h = (float *) malloc(size);
    float *y_h = (float *) malloc(size);
    float *z_h = (float *) malloc(size);
    for(int i = 0;i < N; i++){
        x_h[i] = a_value;
        y_h[i] = b_value;
    }

    /* set init device memory  */
    float *x_d,*y_d,*z_d;
    hipMalloc(&x_d,size); /* here change the ptr position,  so should pass the ptr address */
    hipMalloc((void **) &y_d,size);
    hipMalloc((void **) &z_d,size);

    /* copy the host data to the cuda device */
    hipMemcpy(x_d,x_h,size,hipMemcpyHostToDevice);
    hipMemcpy(y_d,y_h,size,hipMemcpyHostToDevice);

    /* calculate on the device should be dived */
    const int block_size = 128;
    const int grid_size = (N-1) / block_size + 1; /* the common type of grid size  */
    vec_add<<<grid_size,block_size>>>(x_d,y_d,z_d,N);

    /* move the device result to the host */
    hipMemcpy(z_h,z_d,size,hipMemcpyDeviceToHost);
    check(z_h,N);

    free(x_h);
    free(y_h);
    free(z_h);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    return 0;
}