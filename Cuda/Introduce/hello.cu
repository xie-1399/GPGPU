


#include <hip/hip_runtime.h>
#include <stdio.h>
/* write the simple kernel function to print at GPU */

__global__ void helloFromGPU(void){
    printf("Hello World from GPU!\n");
}


int main(){
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1,10>>>(); /* 10 threads are called to execute the func */
    hipDeviceReset(); /* clear all resources about the Device */
    return 0;
}

/*
    Hello World from CPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
    Hello World from GPU!
 */